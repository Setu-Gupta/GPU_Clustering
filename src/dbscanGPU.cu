#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>

#include <thrust/scan.h>
#include <thrust/execution_policy.h>

#include "GPUCluster.h"
__global__ void  kernel_getV(float* x,float* y,unsigned int n,int* V,unsigned char* core,int minPts,float R){
   
    int tx=threadIdx.x+blockDim.x*blockIdx.x;
    // printf("%f\n",R);
   if(tx<n){
    V[tx]=0;
    core[tx]=0;
    float px=x[tx];
    float py=y[tx];
    int count=0;
    for(int i=0;i<n;i++){
        float dist=sqrt(pow(px-x[i],2)+pow(py-y[i],2));
        if(dist<=R && dist>0){
            V[tx]=count+1;
            count++;
        }
    }
    if(count>=minPts){
        core[tx]=1;
    }
    }
}

__global__ void kernel_getEdges(float* x,float* y,unsigned int n,int* V,int* indices,int* edges,float R) {
    int tx=threadIdx.x+blockDim.x*blockIdx.x;
    if(tx<n){
        int count =0;
        float px=x[tx];
        float py=y[tx];
        for(int i=0;i<n;i++){
            float dist=sqrt(pow(px-x[i],2)+pow(py-y[i],2));
            // printf("%f\n",dist);
            if(dist<=R && dist>0){
                edges[indices[tx]+count]=i;
                count++;
            }

        }
    }
    
}
__global__ void kernel_init(unsigned char* Fa,int size){
    int tx=threadIdx.x+blockDim.x*blockIdx.x;
    if(tx<size){
        Fa[tx]=0;
    }
}


__global__ void kernel_bfs_child(int* V,int* indices,int* edges,unsigned char* Fa,unsigned char* Xa,int* workToDo,unsigned int n){
    int tx=threadIdx.x+blockDim.x*blockIdx.x;

    if(tx<n){
    if(Fa[tx]==1){

        Fa[tx]=0;Xa[tx]=1;
        for(int i=0;i<V[tx];i++){
            int neigh=edges[indices[tx]+i];
            if(Xa[neigh]==0){

                Fa[neigh]=1;
                *workToDo=1;
            }


            }
        }
    }
}
__global__ void kernel_updateVisited(unsigned char* visited,unsigned char* Xa,unsigned int* map,int cluster,unsigned int n){
    int tx=threadIdx.x + blockDim.x*blockIdx.x;
    if(tx<n){
        map[tx]=255;
        if(Xa[tx]==1){
            visited[tx]=1;
            map[tx]=cluster;
        }
    }
}
__global__ void kernel_parent_bfs(int n,int* V,int* indices,int* edges,unsigned char* core,unsigned char *Fa,unsigned char* Xa, unsigned int* map,unsigned char* visited,int* workToDo){
    
    kernel_init<<<max(1,n/1024),min(n,1024)>>>(visited,n);
    hipDeviceSynchronize();

    int cluster=1;
    for(int v=0;v<n;v++){

        if(core[v]==1 && visited[v]==0){

            visited[v]=1;
            kernel_init<<<max(1,n/1024),min(n,1024)>>>(Fa,n);
            kernel_init<<<max(1,n/1024),min(n,1024)>>>(Xa,n);
            hipDeviceSynchronize();
            Fa[v]=1;

            *workToDo=1;
            while(*workToDo==1){
                *workToDo=0;
                kernel_bfs_child<<<max(1,n/1024),min(n,1024)>>>(V,indices,edges,Fa,Xa,workToDo,n);
                hipDeviceSynchronize();

            }
            kernel_updateVisited<<<max(1,n/1024),min(n,1024)>>>(visited,Xa,map,cluster,n);
            hipDeviceSynchronize();
            cluster++;
        }
    }

}

float dbscanGPU(float* x, float* y,	unsigned int* map,unsigned int n,int minPts,float R){
    // float x[12]={1,2,3,4,5,6,7,8,20,21,22,89};
    // float y[12]={0,0,0,0,0,0,0,0,0,0,0,0};
    // float R=1;
    // int minPts=2;
    // int n=12;
    struct timespec start_gpu, end_gpu;
	float msecs_gpu;
	clock_gettime(CLOCK_MONOTONIC, &start_gpu);
    int* d_V;
    unsigned char* d_core;
    float* d_x;
    float* d_y;
    hipMalloc((void**)&d_V, sizeof(int)*n);
    hipMalloc((void**)&d_core, sizeof(unsigned char)*n );
    hipMalloc((void**)&d_x, sizeof(float)*n);
    hipMalloc((void**)&d_y, sizeof(float)*n);
    
    
    hipMemcpy(d_x, x, sizeof(float)*n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float)*n, hipMemcpyHostToDevice);
    
    kernel_getV<<<max(1,n/1024),min(n,1024)>>>(d_x,d_y,n,d_V,d_core,minPts,R);
    hipDeviceSynchronize();
    int* V= (int* )malloc(sizeof(int)*n);
    hipMemcpy(V, d_V, sizeof(int)*n, hipMemcpyDeviceToHost);
    
    
    int* indices=(int* )malloc(sizeof(int)*n);
    thrust::exclusive_scan(thrust::host, V, V + n, indices, 0);

    int numEdges=V[n-1]+indices[n-1];
   
    int* edges= (int* ) malloc(sizeof(int)*(numEdges));
    int* d_indices;
    int* d_edges;
    hipMalloc((void**)&d_indices, sizeof(int)*n);
    hipMalloc((void**)&d_edges, sizeof(int)*numEdges);
    
    
    hipMemcpy(d_indices, indices , sizeof(int)*n, hipMemcpyHostToDevice);

    kernel_getEdges<<<max(1,n/1024),min(n,1024)>>>(d_x,d_y,n,d_V,d_indices,d_edges,R);
    hipDeviceSynchronize();
    hipMemcpy(edges, d_edges, sizeof(int)*numEdges, hipMemcpyDeviceToHost);
    

    unsigned int* d_map;
    unsigned char* Fa;unsigned char* Xa;
    hipMalloc((void**)&d_map, sizeof(unsigned int)*n);
    hipMalloc((void**)&Fa, sizeof(unsigned char)*n);
    hipMalloc((void**)&Xa, sizeof(unsigned char)*n);

    unsigned char* visited;
    hipMalloc((void**)&visited, sizeof(unsigned char)*n);
    
    int* d_workToDo;
    hipMalloc((void**)&d_workToDo,sizeof(int));
    
    kernel_parent_bfs<<<1,1>>>(n,d_V,d_indices,d_edges,d_core,Fa,Xa,d_map,visited,d_workToDo);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC, &end_gpu);
	msecs_gpu = 1000.0 * (end_gpu.tv_sec - start_gpu.tv_sec) + (end_gpu.tv_nsec - start_gpu.tv_nsec)/1000000.0;
    // unsigned int* map=(unsigned int*)malloc(sizeof(unsigned int)*n);
    hipMemcpy(map, d_map, sizeof(unsigned int)*n, hipMemcpyDeviceToHost);

    hipFree(d_edges);
    hipFree(d_indices);
    hipFree(d_V);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_core);
    hipFree(Xa);
    hipFree(Fa);
    hipFree(visited);

    free(edges);
    free(indices);
    free(V);

    

    return msecs_gpu;

}